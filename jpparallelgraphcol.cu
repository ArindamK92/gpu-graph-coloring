#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#include <time.h>

#define NO_COLOR 0
#define MIN_COLOR -1
#define MAX_COLOR 1

struct new_csr_graph{
	int v_count,*A, *IA, *color;
};

__global__
void init_kernel(int *d_color, float *d_node_val, hiprandState* state, unsigned long seed, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count){
		hiprand_init ( seed, vertex_id, 0, &state[vertex_id] );
		d_node_val[vertex_id]=hiprand_uniform(state+vertex_id);
		d_color[vertex_id]=NO_COLOR;
	}
}
__global__
void random_generate(float *d_node_val, hiprandState* state, unsigned long seed, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count){
		hiprand_init ( seed, vertex_id, 0, &state[vertex_id] );
		d_node_val[vertex_id]=hiprand_uniform(state+vertex_id);
	}
}
__global__
void minmax_kernel(int *d_A, int *d_IA, int *d_color, float *d_node_val, char *d_color_code, char *d_cont, char *d_change, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count && d_color[vertex_id]==NO_COLOR){
		int total=d_IA[vertex_id+1];
		float curr_node_val=d_node_val[vertex_id];
		float edge_node_val;
		char is_min=1, is_max=1;
		for(int i=d_IA[vertex_id];i<total;i++){
			if(d_color[d_A[i]]!=NO_COLOR){
				//if this adjacent vertex is already colored then continue
				continue;
			}
			edge_node_val=d_node_val[d_A[i]];
			if(edge_node_val<=curr_node_val){
				is_min=0;
			}
			if(edge_node_val>=curr_node_val){
				is_max=0;
			}
		}
		if(is_min){
			d_color_code[vertex_id]=MIN_COLOR;
			*d_change=1;
		}
		else if(is_max){
			d_color_code[vertex_id]=MAX_COLOR;
			*d_change=1;
		}
		else{
			d_color_code[vertex_id]=NO_COLOR;
			*d_cont=1;
		}
	}
}
__global__
void color_kernel(int *d_color, char *d_color_code, int curr_color, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count && d_color[vertex_id]==NO_COLOR){
		if(d_color_code[vertex_id]==MIN_COLOR){
			d_color[vertex_id]=curr_color;
		}
		else if(d_color_code[vertex_id]==MAX_COLOR){
			d_color[vertex_id]=curr_color+1;
		}
	}
}
void assign_color(struct new_csr_graph *input_graph){
	int cur_color=NO_COLOR+1;
	char cont=1, change;
	int *d_A, *d_IA, *d_color;
	char *d_cont, *d_change, *d_color_code;
	float *d_node_val;
	hipMalloc((void **)&d_A,input_graph->IA[input_graph->v_count]*sizeof(int));
	hipMalloc((void **)&d_IA,(input_graph->v_count+1)*sizeof(int));
	hipMalloc((void **)&d_color,input_graph->v_count*sizeof(int));
	hipMalloc((void **)&d_cont,sizeof(char));
	hipMalloc((void **)&d_change,sizeof(char));
	hipMalloc((void **)&d_color_code,input_graph->v_count*sizeof(char));
	hipMalloc((void **)&d_node_val,input_graph->v_count*sizeof(float));

	hipMemcpy(d_A,input_graph->A,input_graph->IA[input_graph->v_count]*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_IA,input_graph->IA,(input_graph->v_count+1)*sizeof(int),hipMemcpyHostToDevice);


	hiprandState* d_states;
	hipMalloc((void **)&d_states, input_graph->v_count * sizeof(hiprandState));
	init_kernel<<<ceil(input_graph->v_count/256.0),256>>>(d_color, d_node_val, d_states, time(NULL), input_graph->v_count);
	hipFree(d_states);

	int rand_ver=1;
	while(cont){
		cont=0;
		change=0;
		hipMemcpy(d_cont,&cont,sizeof(char),hipMemcpyHostToDevice);
		hipMemcpy(d_change,&change,sizeof(char),hipMemcpyHostToDevice);
		minmax_kernel<<<ceil(input_graph->v_count/256.0),256>>>(d_A, d_IA, d_color, d_node_val, d_color_code, d_cont, d_change, input_graph->v_count);
		color_kernel<<<ceil(input_graph->v_count/256.0),256>>>(d_color, d_color_code, cur_color, input_graph->v_count);
		hipMemcpy(&cont,d_cont,sizeof(char),hipMemcpyDeviceToHost);
		hipMemcpy(&change,d_change,sizeof(char),hipMemcpyDeviceToHost);
		if(cont && !change){
			hipMalloc((void **)&d_states, input_graph->v_count * sizeof(hiprandState));
			random_generate<<<ceil(input_graph->v_count/256.0),256>>>(d_node_val, d_states, time(NULL)+rand_ver++, input_graph->v_count);
			hipFree(d_states);
		}
		else{
			cur_color+=2;
		}
	}
	hipFree(d_A);
	hipFree(d_IA);
	hipFree(d_cont);
	hipFree(d_node_val);
	hipMemcpy(input_graph->color,d_color,input_graph->v_count*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_color);
}
int init_input_graph(struct new_csr_graph *input_graph, char *file_name){
	struct edge{
		int vertex1, vertex2;
	}*edge_list;
	FILE *file_pointer ;
	// in read mode using "r" attribute
	file_pointer = fopen(file_name, "r") ;
	if ( file_pointer == NULL )
	{
		return 1;
	}
	char new_line_flag=1, line_type=0, c;
	int param1=0, param2=0;
	int phase=1, edge_id=0, e_count, i;
	for (c = getc(file_pointer); c != EOF; c = getc(file_pointer))
	{
		if(c=='\n'){
			new_line_flag=1;
			if(line_type=='p'){
				input_graph->v_count=param1;
				e_count=param2;
				input_graph->IA=(int *)malloc((param1+1)*sizeof(int));
				input_graph->A=(int *)malloc(param2*2*sizeof(int));
				input_graph->color=(int *)malloc(param1*sizeof(int));
				for(i=0;i<=param1;i++){
					input_graph->IA[i]=0;
				}
				edge_list=(struct edge *)malloc(param2*sizeof(struct edge));
			}
			else if(line_type=='e'){
				edge_list[edge_id].vertex1=param1-1;
				edge_list[edge_id].vertex2=param2-1;
				input_graph->IA[param1]++;
				input_graph->IA[param2]++;
				edge_id++;
			}

			param1=0, param2=0;
			line_type=0;
			continue;
		}
		if(new_line_flag){
			line_type=c;
			phase=1;
			new_line_flag=0;
			continue;
		}
		if(line_type=='e'){
			switch(phase){
			case 1:
				if(c>='0' && c<='9'){
					param1=c-'0';
					phase++;
				}
				break;
			case 2:
				if(c>='0' && c<='9'){
					param1=param1*10+c-'0';
				}
				else{
					phase++;
				}
				break;
			case 3:
				if(c>='0' && c<='9'){
					param2=param2*10+c-'0';
				}
				else{
					phase++;
				}
				break;
			}
		}

		else if(line_type=='p'){
			switch(phase){
			case 1:
				if(c>='0' && c<='9'){
					param1=c-'0';
					phase++;
				}
				break;
			case 2:
				if(c>='0' && c<='9'){
					param1=param1*10+c-'0';
				}
				else{
					phase++;
				}
				break;
			case 3:
				if(c>='0' && c<='9'){
					param2=param2*10+c-'0';
				}
				else{
					phase++;
				}
				break;
			}
		}
	}
	fclose(file_pointer) ;
	if(!new_line_flag && line_type=='e'){
		edge_list[edge_id].vertex1=param1-1;
		edge_list[edge_id].vertex2=param2-1;
		input_graph->IA[param1]++;
		input_graph->IA[param2]++;
	}
	int *vertex_p=(int *)malloc(input_graph->v_count*sizeof(int));
	for(i=0;i<input_graph->v_count;i++){
		input_graph->IA[i+1]+=input_graph->IA[i];
		vertex_p[i]=0;
	}
	for(edge_id=0;edge_id<e_count;edge_id++){
		input_graph->A[input_graph->IA[edge_list[edge_id].vertex1]+(vertex_p[edge_list[edge_id].vertex1]++)]=edge_list[edge_id].vertex2;
		input_graph->A[input_graph->IA[edge_list[edge_id].vertex2]+(vertex_p[edge_list[edge_id].vertex2]++)]=edge_list[edge_id].vertex1;
	}
	free(edge_list);
	free(vertex_p);
	return 0;
}
int validate_coloring(struct new_csr_graph *input_graph){
	for(int i=0;i<input_graph->v_count;i++){
		for(int j=input_graph->IA[i];j<input_graph->IA[i+1];j++){
			if(input_graph->color[i]==input_graph->color[input_graph->A[j]]){
				return 0;
			}
		}
	}
	return 1;
}
int count_colors(struct new_csr_graph *input_graph){
	int max_color_used=0;
	for(int i=0;i<input_graph->v_count;i++){
		max_color_used=max_color_used>input_graph->color[i]?max_color_used:input_graph->color[i];
	}
	char *color_used=(char *)malloc(sizeof(char)*max_color_used);
	int total_colors=0;
	for(int i=0;i<max_color_used;i++){
		color_used[i]=0;
	}
	for(int i=0;i<input_graph->v_count;i++){
		color_used[input_graph->color[i]]=1;
	}
	for(int i=0;i<max_color_used;i++){
		if(color_used[i]==1){
			total_colors++;
		}
	}
	return total_colors;
}
int main(){
	struct new_csr_graph input_graph;
	init_input_graph(&input_graph, "input.txt");

	clock_t start, end;
	double cpu_time_used;
	start = clock();

	assign_color(&input_graph);

	end = clock();
	cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

	printf("\ntime taken:%f",cpu_time_used);
	if(!validate_coloring(&input_graph)){
		printf("\nInvalid coloring!");
		return 0;
	}
	printf("\nNo. of colors used:%d",count_colors(&input_graph));
	printf("\nresult coloring:");
	for(int i=0;i<input_graph.v_count;i++){
		printf("%d ",input_graph.color[i]);
	}
}
