#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>

#define NO_COLOR 0
#define MIN_COLOR -1
#define MAX_COLOR 1

struct new_csr_graph{
	int v_count,*A, *IA, *color;
};

__global__
void init_kernel(int *d_color, float *d_node_val, int *d_IA, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count){
		d_node_val[vertex_id]=d_IA[vertex_id+1]-d_IA[vertex_id];
		d_color[vertex_id]=NO_COLOR;
	}
}
__global__
void random_generate(float *d_node_val, hiprandState* state, unsigned long seed, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count){
		hiprand_init ( seed, vertex_id, 0, &state[vertex_id] );
		d_node_val[vertex_id]=hiprand_uniform(state+vertex_id);
	}
}
__global__
void minmax_kernel(int *d_A, int *d_IA, int *d_color, float *d_node_val, char *d_color_code, char *d_cont, char *d_change, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count && d_color[vertex_id]==NO_COLOR){
		int total=d_IA[vertex_id+1];
		float curr_node_val=d_node_val[vertex_id];
		float edge_node_val;
		char is_min=1, is_max=1;
		for(int i=d_IA[vertex_id];i<total;i++){
			edge_node_val=d_node_val[d_A[i]];
			if(edge_node_val<=curr_node_val){
				is_min=0;
			}
			if(edge_node_val>=curr_node_val){
				is_max=0;
			}
		}
		if(is_min){
			d_color_code[vertex_id]=MIN_COLOR;
			*d_change=1;
		}
		else if(is_max){
			d_color_code[vertex_id]=MAX_COLOR;
			*d_change=1;
		}
		else{
			d_color_code[vertex_id]=NO_COLOR;
			*d_cont=1;
		}
	}
}
__global__
void color_kernel(int *d_color, char *d_color_code, int curr_color, int v_count){
	int vertex_id=blockIdx.x*blockDim.x+threadIdx.x;
	if(vertex_id<v_count && d_color[vertex_id]==NO_COLOR){
		if(d_color_code[vertex_id]==MIN_COLOR){
			d_color[vertex_id]=curr_color;
		}
		else if(d_color_code[vertex_id]==MAX_COLOR){
			d_color[vertex_id]=curr_color+1;
		}
	}
}
void assign_color(struct new_csr_graph *input_graph){
	int cur_color=NO_COLOR+1;
	char cont=1, change;
	int *d_A, *d_IA, *d_color;
	char *d_cont, *d_change, *d_color_code;
	float *d_node_val;
	hipMalloc((void **)&d_A,input_graph->IA[input_graph->v_count]*sizeof(int));
	hipMalloc((void **)&d_IA,(input_graph->v_count+1)*sizeof(int));
	hipMalloc((void **)&d_color,input_graph->v_count*sizeof(int));
	hipMalloc((void **)&d_cont,sizeof(char));
	hipMalloc((void **)&d_change,sizeof(char));
	hipMalloc((void **)&d_color_code,input_graph->v_count*sizeof(char));
	hipMalloc((void **)&d_node_val,input_graph->v_count*sizeof(float));

	hipMemcpy(d_A,input_graph->A,input_graph->IA[input_graph->v_count]*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_IA,input_graph->IA,(input_graph->v_count+1)*sizeof(int),hipMemcpyHostToDevice);

	init_kernel<<<ceil(input_graph->v_count/256.0),256>>>(d_color, d_node_val, d_IA, input_graph->v_count);

	while(cont){
		cont=0;
		change=0;
		hipMemcpy(d_cont,&cont,sizeof(char),hipMemcpyHostToDevice);
		hipMemcpy(d_change,&change,sizeof(char),hipMemcpyHostToDevice);
		minmax_kernel<<<ceil(input_graph->v_count/256.0),256>>>(d_A, d_IA, d_color, d_node_val, d_color_code, d_cont, d_change, input_graph->v_count);
		color_kernel<<<ceil(input_graph->v_count/256.0),256>>>(d_color, d_color_code, cur_color, input_graph->v_count);
		hipMemcpy(&cont,d_cont,sizeof(char),hipMemcpyDeviceToHost);
		hipMemcpy(&change,d_change,sizeof(char),hipMemcpyDeviceToHost);
		if(cont && !change){
			hiprandState* d_states;
			hipMalloc((void **)&d_states, input_graph->v_count * sizeof(hiprandState));
			random_generate<<<ceil(input_graph->v_count/256.0),256>>>(d_node_val, d_states, time(NULL), input_graph->v_count);
			hipFree(d_states);
		}
		else{
			cur_color+=2;
		}
	}
	hipFree(d_A);
	hipFree(d_IA);
	hipFree(d_cont);
	hipFree(d_node_val);
	hipMemcpy(input_graph->color,d_color,input_graph->v_count*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_color);
}

void init_input_graph(struct new_csr_graph *input_graph){
	input_graph->v_count=8;
	input_graph->A=(int *)malloc(sizeof(int)*16);
	input_graph->IA=(int *)malloc(sizeof(int)*9);
	input_graph->color=(int *)malloc(sizeof(int)*input_graph->v_count);
	input_graph->A[0]=1;
	input_graph->A[1]=2;
	input_graph->A[2]=0;
	input_graph->A[3]=2;
	input_graph->A[4]=0;
	input_graph->A[5]=1;
	input_graph->A[6]=3;
	input_graph->A[7]=2;
	input_graph->A[8]=4;
	input_graph->A[9]=3;
	input_graph->A[10]=5;
	input_graph->A[11]=6;
	input_graph->A[12]=7;
	input_graph->A[13]=4;
	input_graph->A[14]=4;
	input_graph->A[15]=4;

	input_graph->IA[0]=0;
	input_graph->IA[1]=2;
	input_graph->IA[2]=4;
	input_graph->IA[3]=7;
	input_graph->IA[4]=9;
	input_graph->IA[5]=13;
	input_graph->IA[6]=14;
	input_graph->IA[7]=15;
	input_graph->IA[8]=16;
}
int main(){
	struct new_csr_graph input_graph;
	init_input_graph(&input_graph);
	assign_color(&input_graph);
	printf("\nresult coloring:");
	for(int i=0;i<input_graph.v_count;i++){
		printf("%d ",input_graph.color[i]);
	}
}
